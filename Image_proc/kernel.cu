#include "hip/hip_runtime.h"
﻿
#include <hip/hip_runtime.h>
#include ""
#include  <time.h>
#include  <stdlib.h>
#include  <stdio.h>
#include <iostream>
#include  <string.h>
#include  <math.h>
#include  <hip/hip_runtime.h>
#include  <ctime>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image.h"
#include "stb_image_write.h"

using namespace std;
//Global parameters to be set according to the image to be processed
int width, height;
int mask[3][3] = {{1,2,1},
                  {2,3,2},
                  {1,2,1}
                  };

//Masking function for Host
int getPix(unsigned char* arr, int row, int col) {
    int Pix = 0;
    //Traversing all the 9 pixels in the surroundings
    for (int i = -1;i < 2;i++) {
        for (int j = -1;j < 2;j++) {
            Pix += arr[((row + i) * width) + (col + j)] * mask[i+1][j+1];
        }
    }
    //Normalising the weight
    return Pix / 15;
}

//To be called by the Host and run on the host
//Serial computation
void Host_blur(unsigned char* arr, unsigned char* Img) {
    int offset = 2 * width;
    for ( int row = 2;row < height - 2;row++) {
        for ( int col = 2; col < width - 2; col++) {
            Img[offset + col] = getPix(arr, row, col);
        }
        offset += width;
    }
    return;
}

__global__ void Device_blur(unsigned char* arr, unsigned char* Img, int width, int height) {
    int row=blockIdx.x+blockDim.x + threadIdx.x;
    int col= blockIdx.y + blockDim.y + threadIdx.y;
    if ((row < 0) || (col < 0) || (row >= height ) || (col >= width)) {
        return;
    }
    if ((row < 2) || (col < 2) || (row >= height - 2) || (col >= width - 2)) {
        Img[row * width + col] = 'A';
        return;
    }
    
    int mask[3][3] = { {1,2,1},
                  {2,3,2},
                  {1,2,1}};

    int Pix = 0;
    
    for (int i = -1;i < 2;i++) {
        for (int j = -1;j < 2;j++) {
            Pix += arr[((row + i) * width) + (col + j)] * mask[i + 1][j + 1];
        }
    }

    Img[row * width + col] = Pix / 15;
}

int main(int argc, char** argv) {
    //Array pointers
    unsigned char  *Host_Raw=NULL   ,* Device_Raw=NULL, *Host_Final_Img;

    //Fetching the images
    int channels;
    unsigned char *A = stbi_load("Kakashi.png", &width, &height, &channels, 0);

    Host_Raw = (unsigned char*) malloc(width * height);
    Host_Final_Img = (unsigned char*)malloc(width * height);

    for (int i = 0;i < width * height;i++) {
        Host_Raw[i] = (A[i * channels] + A[i * channels + 1] + A[i * channels + 2])/3;
    }

    int ImageSize = sizeof(unsigned char) * width * height;

    //Host_Final_Img = (unsigned char*)malloc(ImageSize);
    stbi_write_png("kaka_gray.png", width, height, 1, Host_Raw, width );

    //return 0;
//    hipMalloc((void**)&Device_Raw, ImageSize);
//    hipMalloc((void**)&Device_Final_Img, ImageSize);

    //************// Setup Work //**************************//
    
    //*********// Host //******************//
    clock_t starttime, endtime, diff;
    starttime = clock();
    Host_blur(Host_Raw, Host_Final_Img);
    endtime = clock();
    diff = endtime - starttime;
    double interval = diff / ((double)CLOCKS_PER_SEC);
    cout << "CPU executed in milisec: " << interval*1000<<endl;

    stbi_write_png("kaka_proc.png", width, height, 1, Host_Final_Img, width);


    //cutSavePGMub(Host_Res_Path, Host_Final_Img, width, height);
   //****// End of host processing //*************//
   
    int x;
    cout << "end!";
    cin >> x;
}



